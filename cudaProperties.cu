#include "stdio.h"

int main(void){
   hipDeviceProp_t prop;
   int count;
   hipGetDeviceCount(&count);

   for(int i = 0 ; i < count ; i++){
     hipGetDeviceProperties(&prop, i);
     printf("Name: %s\n", prop.name);
     printf("Compute capability: %d.%d\n", prop.major, prop.minor);
     printf("Clock rate: %d\n", prop.clockRate);
     printf("Total global mem: %d\n", prop.totalGlobalMem);
     printf("Max threads per block: %d\n",prop.maxThreadsPerBlock);


   }

}
