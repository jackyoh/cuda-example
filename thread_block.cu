
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
//#include "conio.h"

__global__ void what_is_my_id(int * block, int * thread, int * wrap, int * calc_thread){
  const int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  block[thread_idx] = blockIdx.x;
  thread[thread_idx] = threadIdx.x;

  wrap[thread_idx] = threadIdx.x / warpSize;
  calc_thread[thread_idx] = thread_idx;
}

#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))

int cpu_block[ARRAY_SIZE];
int cpu_thread[ARRAY_SIZE];
int cpu_warp[ARRAY_SIZE];
int cpu_calc_thread[ARRAY_SIZE];

int main(void){
  int num_blocks = 2;
  int num_threads = 64;
  //char ch;

  int * gpu_block;
  int * gpu_thread;
  int * gpu_warp;
  int * gpu_calc_thread;

  int i;
  
  hipMalloc((void **)&gpu_block, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_warp, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);

  what_is_my_id<<<num_blocks, num_threads>>>(gpu_block, gpu_thread, gpu_warp, gpu_calc_thread);
 
  hipMemcpy(cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(cpu_warp, gpu_warp, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

  hipFree(gpu_block);
  hipFree(gpu_thread);
  hipFree(gpu_warp);
  hipFree(gpu_calc_thread);

  for(i=0;i<ARRAY_SIZE;i++){
     printf("Calculated Thread: %3u - Block: %2u - Warp %2u - Thread %3u\n", cpu_calc_thread[i], cpu_block[i], cpu_warp[i], cpu_thread[i]);
  }
  //ch = getch();
} 
